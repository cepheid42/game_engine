#include "em_kernel.cuh"

#include <iostream>


template struct cudaClass<Thingy>;

#define cudaChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
   if (code != hipSuccess) {
      std::cout << "[" << file << ":" << line << "] GPU Error: " << hipGetErrorString(code) << std::endl;
      if (abort) { exit(code); }
   }
}

template<typename T>
void cudaClass<T>::run() const {
   float* result;
   cudaChk(hipMallocManaged(&result, 32 * sizeof(float)));
   cudaChk(hipDeviceSynchronize());


   kernel<T><<<1, 32>>>(t, 2.0f, result);
   cudaChk(hipDeviceSynchronize());

   for (int i = 0; i < 32; i++) {
      std::cout << result[i] << ", ";
   }
   std::cout << std::endl;

   cudaChk(hipFree(result));
   cudaChk(hipDeviceSynchronize());
}


